#include "hip/hip_runtime.h"
/*
 * Scalar Born wave equation propagator (CUDA implementation)
 */

/*
 * This file contains the CUDA implementation of the scalar Born wave equation
 * propagator. It is compiled multiple times with different options
 * to generate a set of functions that can be called from Python.
 * The options are specified by the following macros:
 *  * DW_ACCURACY: The order of accuracy of the spatial finite difference
 *    stencil. Possible values are 2, 4, 6, and 8.
 *  * DW_DTYPE: The floating point type to use for calculations. Possible
 *    values are float and double.
 */

/*
 * For a description of the method, see the C implementation in
 * scalar_born.c and scalar.c. This file implements the same functionality,
 * but for execution on a GPU using CUDA.
 */

#include <stdio.h>

#include <cstdint>

#include "common.h"

#define CAT_I(name, accuracy, dtype, device) \
  scalar_born_iso_##accuracy##_##dtype##_##name##_##device
#define CAT(name, accuracy, dtype, device) CAT_I(name, accuracy, dtype, device)
#define FUNC(name) CAT(name, DW_ACCURACY, DW_DTYPE, DW_DEVICE)

#define WFC(dy, dx) wfc[i + dy * nx + dx]
#define WFCSC(dy, dx) wfcsc[i + dy * nx + dx]
#define AY_PSIY(dy, dx) ay[y + dy] * psiy[i + dy * nx + dx]
#define AX_PSIX(dy, dx) ax[x + dx] * psix[i + dy * nx + dx]
#define AY_PSIYSC(dy, dx) ay[y + dy] * psiysc[i + dy * nx + dx]
#define AX_PSIXSC(dy, dx) ax[x + dx] * psixsc[i + dy * nx + dx]
#define V(dy, dx) v_shot[j + dy * nx + dx]
#define VDT2(dy, dx) V(dy, dx) * dt2
#define V2DT2(dy, dx) V(dy, dx) * V(dy, dx) * dt2
#define SCATTER(dy, dx) scatter_shot[j + dy * nx + dx]
#define V2DT2_WFC(dy, dx)        \
  (V2DT2(dy, dx) * WFC(dy, dx) + \
   2 * VDT2(dy, dx) * SCATTER(dy, dx) * WFCSC(dy, dx))
#define V2DT2_WFCSC(dy, dx) V2DT2(dy, dx) * WFCSC(dy, dx)
#define UT_TERMY1(dy, dx)                                                      \
  (dbydy[y + dy] * ((1 + by[y + dy]) *                                         \
                        (V2DT2(dy, dx) * WFC(dy, dx) +                         \
                         2 * VDT2(dy, dx) * SCATTER(dy, dx) * WFCSC(dy, dx)) + \
                    by[y + dy] * zetay[i + dy * nx]) +                         \
   by[y + dy] * psiy[i + dy * nx])
#define UT_TERMX1(dy, dx)                                                      \
  (dbxdx[x + dx] * ((1 + bx[x + dx]) *                                         \
                        (V2DT2(dy, dx) * WFC(dy, dx) +                         \
                         2 * VDT2(dy, dx) * SCATTER(dy, dx) * WFCSC(dy, dx)) + \
                    bx[x + dx] * zetax[i + dx]) +                              \
   bx[x + dx] * psix[i + dx])
#define UT_TERMY2(dy, dx)                                                     \
  ((1 + by[y + dy]) *                                                         \
   ((1 + by[y + dy]) * (V2DT2(dy, dx) * WFC(dy, dx) +                         \
                        2 * VDT2(dy, dx) * SCATTER(dy, dx) * WFCSC(dy, dx)) + \
    by[y + dy] * zetay[i + dy * nx]))
#define UT_TERMX2(dy, dx)                                                     \
  ((1 + bx[x + dx]) *                                                         \
   ((1 + bx[x + dx]) * (V2DT2(dy, dx) * WFC(dy, dx) +                         \
                        2 * VDT2(dy, dx) * SCATTER(dy, dx) * WFCSC(dy, dx)) + \
    bx[x + dx] * zetax[i + dx]))
#define PSIY_TERM(dy, dx)                                                    \
  ((1 + by[y + dy]) * (V2DT2(dy, dx) * WFC(dy, dx) +                         \
                       2 * VDT2(dy, dx) * SCATTER(dy, dx) * WFCSC(dy, dx)) + \
   by[y + dy] * zetay[i + dy * nx])
#define PSIX_TERM(dy, dx)                                                    \
  ((1 + bx[x + dx]) * (V2DT2(dy, dx) * WFC(dy, dx) +                         \
                       2 * VDT2(dy, dx) * SCATTER(dy, dx) * WFCSC(dy, dx)) + \
   bx[x + dx] * zetax[i + dx])
#define UTSC_TERMY1(dy, dx)                                             \
  ((dbydy[y + dy] * ((1 + by[y + dy]) * V2DT2(dy, dx) * WFCSC(dy, dx) + \
                     by[y + dy] * zetaysc[i + dy * nx]) +               \
    by[y + dy] * psiysc[i + dy * nx]))
#define UTSC_TERMX1(dy, dx)                                             \
  ((dbxdx[x + dx] * ((1 + bx[x + dx]) * V2DT2(dy, dx) * WFCSC(dy, dx) + \
                     bx[x + dx] * zetaxsc[i + dx]) +                    \
    bx[x + dx] * psixsc[i + dx]))
#define UTSC_TERMY2(dy, dx)                                               \
  ((1 + by[y + dy]) * ((1 + by[y + dy]) * V2DT2(dy, dx) * WFCSC(dy, dx) + \
                       by[y + dy] * zetaysc[i + dy * nx]))
#define UTSC_TERMX2(dy, dx)                                               \
  ((1 + bx[x + dx]) * ((1 + bx[x + dx]) * V2DT2(dy, dx) * WFCSC(dy, dx) + \
                       bx[x + dx] * zetaxsc[i + dx]))
#define PSIYSC_TERM(dy, dx)                           \
  ((1 + by[y + dy]) * V2DT2(dy, dx) * WFCSC(dy, dx) + \
   by[y + dy] * zetaysc[i + dy * nx])
#define PSIXSC_TERM(dy, dx)                           \
  ((1 + bx[x + dx]) * V2DT2(dy, dx) * WFCSC(dy, dx) + \
   bx[x + dx] * zetaxsc[i + dx])

#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }

namespace {
__constant__ DW_DTYPE dt2;
__constant__ DW_DTYPE rdy;
__constant__ DW_DTYPE rdx;
__constant__ DW_DTYPE rdy2;
__constant__ DW_DTYPE rdx2;
__constant__ int64_t n_shots;
__constant__ int64_t ny;
__constant__ int64_t nx;
__constant__ int64_t nynx;
__constant__ int64_t n_sources_per_shot;
__constant__ int64_t n_sourcessc_per_shot;
__constant__ int64_t n_receivers_per_shot;
__constant__ int64_t n_receiverssc_per_shot;
__constant__ int64_t step_ratio;
__constant__ int64_t pml_y0;
__constant__ int64_t pml_y1;
__constant__ int64_t pml_x0;
__constant__ int64_t pml_x1;
__constant__ bool v_batched;
__constant__ bool scatter_batched;

__global__ void add_sources_both(DW_DTYPE *__restrict const wf,
                                 DW_DTYPE *__restrict const wfsc,
                                 DW_DTYPE const *__restrict const f,
                                 DW_DTYPE const *__restrict const fsc,
                                 int64_t const *__restrict const sources_i) {
  int64_t source_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t shot_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (source_idx < n_sources_per_shot && shot_idx < n_shots) {
    int64_t k = shot_idx * n_sources_per_shot + source_idx;
    if (0 <= sources_i[k]) {
      wf[shot_idx * nynx + sources_i[k]] += f[k];
      wfsc[shot_idx * nynx + sources_i[k]] += fsc[k];
    }
  }
}

__global__ void add_adjoint_sources(DW_DTYPE *__restrict const wf,
                                    DW_DTYPE const *__restrict const f,
                                    int64_t const *__restrict const sources_i) {
  int64_t source_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t shot_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (source_idx < n_receivers_per_shot && shot_idx < n_shots) {
    int64_t k = shot_idx * n_receivers_per_shot + source_idx;
    if (0 <= sources_i[k]) wf[shot_idx * nynx + sources_i[k]] += f[k];
  }
}

__global__ void add_adjoint_sourcessc(
    DW_DTYPE *__restrict const wf, DW_DTYPE const *__restrict const f,
    int64_t const *__restrict const sources_i) {
  int64_t source_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t shot_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (source_idx < n_receiverssc_per_shot && shot_idx < n_shots) {
    int64_t k = shot_idx * n_receiverssc_per_shot + source_idx;
    if (0 <= sources_i[k]) wf[shot_idx * nynx + sources_i[k]] += f[k];
  }
}

__global__ void record_receivers(DW_DTYPE *__restrict const r,
                                 DW_DTYPE const *__restrict const wf,
                                 int64_t const *__restrict receivers_i) {
  int64_t receiver_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t shot_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (receiver_idx < n_receivers_per_shot && shot_idx < n_shots) {
    int64_t k = shot_idx * n_receivers_per_shot + receiver_idx;
    if (0 <= receivers_i[k]) r[k] = wf[shot_idx * nynx + receivers_i[k]];
  }
}

__global__ void record_receiverssc(DW_DTYPE *__restrict const r,
                                   DW_DTYPE const *__restrict const wf,
                                   int64_t const *__restrict receivers_i) {
  int64_t receiver_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t shot_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (receiver_idx < n_receiverssc_per_shot && shot_idx < n_shots) {
    int64_t k = shot_idx * n_receiverssc_per_shot + receiver_idx;
    if (0 <= receivers_i[k]) r[k] = wf[shot_idx * nynx + receivers_i[k]];
  }
}

__global__ void record_adjoint_receivers(
    DW_DTYPE *__restrict const r, DW_DTYPE const *__restrict const wf,
    int64_t const *__restrict receivers_i) {
  int64_t receiver_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t shot_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (receiver_idx < n_sources_per_shot && shot_idx < n_shots) {
    int64_t k = shot_idx * n_sources_per_shot + receiver_idx;
    if (0 <= receivers_i[k]) r[k] = wf[shot_idx * nynx + receivers_i[k]];
  }
}

__global__ void record_adjoint_receiverssc(
    DW_DTYPE *__restrict const rsc, DW_DTYPE const *__restrict const wfsc,
    int64_t const *__restrict receivers_i) {
  int64_t receiver_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t shot_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (receiver_idx < n_sourcessc_per_shot && shot_idx < n_shots) {
    int64_t k = shot_idx * n_sourcessc_per_shot + receiver_idx;
    if (0 <= receivers_i[k]) rsc[k] = wfsc[shot_idx * nynx + receivers_i[k]];
  }
}

__global__ void combine_grad(DW_DTYPE *__restrict const grad,
                             DW_DTYPE const *__restrict const grad_shot) {
  int64_t x = blockIdx.x * blockDim.x + threadIdx.x + FD_PAD;
  int64_t y = blockIdx.y * blockDim.y + threadIdx.y + FD_PAD;
  int64_t i = y * nx + x;
  if (y < ny - FD_PAD && x < nx - FD_PAD) {
    int64_t shot_idx;
    for (shot_idx = 0; shot_idx < n_shots; ++shot_idx) {
      grad[i] += grad_shot[shot_idx * nynx + i];
    }
  }
}

__global__ void forward_kernel(
    DW_DTYPE const *__restrict const v,
    DW_DTYPE const *__restrict const scatter,
    DW_DTYPE const *__restrict const wfc, DW_DTYPE *__restrict const wfp,
    DW_DTYPE const *__restrict const psiy,
    DW_DTYPE const *__restrict const psix, DW_DTYPE *__restrict const psiyn,
    DW_DTYPE *__restrict const psixn, DW_DTYPE *__restrict const zetay,
    DW_DTYPE *__restrict const zetax, DW_DTYPE const *__restrict const wfcsc,
    DW_DTYPE *__restrict const wfpsc, DW_DTYPE const *__restrict const psiysc,
    DW_DTYPE const *__restrict const psixsc, DW_DTYPE *__restrict const psiynsc,
    DW_DTYPE *__restrict const psixnsc, DW_DTYPE *__restrict const zetaysc,
    DW_DTYPE *__restrict const zetaxsc, DW_DTYPE *__restrict const w_store,
    DW_DTYPE *__restrict const wsc_store, DW_DTYPE const *__restrict const ay,
    DW_DTYPE const *__restrict const ax, DW_DTYPE const *__restrict const by,
    DW_DTYPE const *__restrict const bx, DW_DTYPE const *__restrict const dbydy,
    DW_DTYPE const *__restrict const dbxdx, bool const v_requires_grad,
    bool const scatter_requires_grad) {
  int64_t x = blockIdx.x * blockDim.x + threadIdx.x + FD_PAD;
  int64_t y = blockIdx.y * blockDim.y + threadIdx.y + FD_PAD;
  if (y < ny - FD_PAD && x < nx - FD_PAD) {
    int64_t shot_idx = blockIdx.z * blockDim.z + threadIdx.z;
    int64_t j = y * nx + x;
    int64_t i = shot_idx * nynx + j;
    DW_DTYPE const *__restrict const v_shot = v_batched ? v + shot_idx * nynx : v;
    DW_DTYPE const *__restrict const scatter_shot = scatter_batched ? scatter + shot_idx * nynx : scatter;
    bool pml_y = y < pml_y0 || y >= pml_y1;
    bool pml_x = x < pml_x0 || x >= pml_x1;
    DW_DTYPE w_sum, wsc_sum;
    if (!pml_y) {
      w_sum = DIFFY2(WFC);
      wsc_sum = DIFFY2(WFCSC);
    } else {
      DW_DTYPE dwfcdy = DIFFY1(WFC);
      DW_DTYPE tmpy =
          ((1 + by[y]) * DIFFY2(WFC) + dbydy[y] * dwfcdy + DIFFY1(AY_PSIY));
      DW_DTYPE dwfcscdy = DIFFY1(WFCSC);
      DW_DTYPE tmpysc = ((1 + by[y]) * DIFFY2(WFCSC) + dbydy[y] * dwfcscdy +
                         DIFFY1(AY_PSIYSC));
      w_sum = (1 + by[y]) * tmpy + ay[y] * zetay[i];
      wsc_sum = (1 + by[y]) * tmpysc + ay[y] * zetaysc[i];
      psiyn[i] = by[y] * dwfcdy + ay[y] * psiy[i];
      zetay[i] = by[y] * tmpy + ay[y] * zetay[i];
      psiynsc[i] = by[y] * dwfcscdy + ay[y] * psiysc[i];
      zetaysc[i] = by[y] * tmpysc + ay[y] * zetaysc[i];
    }
    if (!pml_x) {
      w_sum += DIFFX2(WFC);
      wsc_sum += DIFFX2(WFCSC);
    } else {
      DW_DTYPE dwfcdx = DIFFX1(WFC);
      DW_DTYPE tmpx =
          ((1 + bx[x]) * DIFFX2(WFC) + dbxdx[x] * dwfcdx + DIFFX1(AX_PSIX));
      DW_DTYPE dwfcscdx = DIFFX1(WFCSC);
      DW_DTYPE tmpxsc = ((1 + bx[x]) * DIFFX2(WFCSC) + dbxdx[x] * dwfcscdx +
                         DIFFX1(AX_PSIXSC));
      w_sum += (1 + bx[x]) * tmpx + ax[x] * zetax[i];
      wsc_sum += (1 + bx[x]) * tmpxsc + ax[x] * zetaxsc[i];
      psixn[i] = bx[x] * dwfcdx + ax[x] * psix[i];
      zetax[i] = bx[x] * tmpx + ax[x] * zetax[i];
      psixnsc[i] = bx[x] * dwfcscdx + ax[x] * psixsc[i];
      zetaxsc[i] = bx[x] * tmpxsc + ax[x] * zetaxsc[i];
    }
    wfp[i] = v_shot[j] * v_shot[j] * dt2 * w_sum + 2 * wfc[i] - wfp[i];
    wfpsc[i] = v_shot[j] * v_shot[j] * dt2 * wsc_sum + 2 * wfcsc[i] - wfpsc[i] +
               2 * v_shot[j] * scatter_shot[j] * dt2 * w_sum;
    if (v_requires_grad || scatter_requires_grad) {
      w_store[i] = w_sum;
    }
    if (v_requires_grad) {
      wsc_store[i] = wsc_sum;
    }
  }
}

__global__ void backward_kernel(
    DW_DTYPE const *__restrict const v,
    DW_DTYPE const *__restrict const scatter,
    DW_DTYPE const *__restrict const wfc, DW_DTYPE *__restrict const wfp,
    DW_DTYPE const *__restrict const psiy,
    DW_DTYPE const *__restrict const psix, DW_DTYPE *__restrict const psiyn,
    DW_DTYPE *__restrict const psixn, DW_DTYPE *__restrict const zetay,
    DW_DTYPE *__restrict const zetax, DW_DTYPE *__restrict const zetayn,
    DW_DTYPE *__restrict const zetaxn, DW_DTYPE const *__restrict const wfcsc,
    DW_DTYPE *__restrict const wfpsc, DW_DTYPE const *__restrict const psiysc,
    DW_DTYPE const *__restrict const psixsc, DW_DTYPE *__restrict const psiynsc,
    DW_DTYPE *__restrict const psixnsc, DW_DTYPE *__restrict const zetaysc,
    DW_DTYPE *__restrict const zetaxsc, DW_DTYPE *__restrict const zetaynsc,
    DW_DTYPE *__restrict const zetaxnsc,
    DW_DTYPE const *__restrict const w_store,
    DW_DTYPE const *__restrict const wsc_store,
    DW_DTYPE *__restrict const grad_v, DW_DTYPE *__restrict const grad_scatter,
    DW_DTYPE const *__restrict const ay, DW_DTYPE const *__restrict const ax,
    DW_DTYPE const *__restrict const by, DW_DTYPE const *__restrict const bx,
    DW_DTYPE const *__restrict const dbydy,
    DW_DTYPE const *__restrict const dbxdx, bool const v_requires_grad,
    bool const scatter_requires_grad) {
  int64_t x = blockIdx.x * blockDim.x + threadIdx.x + FD_PAD;
  int64_t y = blockIdx.y * blockDim.y + threadIdx.y + FD_PAD;
  if (y < ny - FD_PAD && x < nx - FD_PAD) {
    int64_t shot_idx = blockIdx.z * blockDim.z + threadIdx.z;
    int64_t j = y * nx + x;
    int64_t i = shot_idx * nynx + j;
    DW_DTYPE const *__restrict const v_shot = v_batched ? v + shot_idx * nynx : v;
    DW_DTYPE const *__restrict const scatter_shot = scatter_batched ? scatter + shot_idx * nynx : scatter;
    bool pml_y = y < pml_y0 || y >= pml_y1;
    bool pml_x = x < pml_x0 || x >= pml_x1;
    wfp[i] =
        (pml_y ? -DIFFY1(UT_TERMY1) + DIFFY2(UT_TERMY2) : DIFFY2(V2DT2_WFC)) +
        (pml_x ? -DIFFX1(UT_TERMX1) + DIFFX2(UT_TERMX2) : DIFFX2(V2DT2_WFC)) +
        2 * wfc[i] - wfp[i];
    wfpsc[i] = (pml_y ? -DIFFY1(UTSC_TERMY1) + DIFFY2(UTSC_TERMY2)
                      : DIFFY2(V2DT2_WFCSC)) +
               (pml_x ? -DIFFX1(UTSC_TERMX1) + DIFFX2(UTSC_TERMX2)
                      : DIFFX2(V2DT2_WFCSC)) +
               2 * wfcsc[i] - wfpsc[i];
    if (pml_y) {
      psiynsc[i] = -ay[y] * DIFFY1(PSIYSC_TERM) + ay[y] * psiysc[i];
      zetaynsc[i] = ay[y] * V2DT2(0, 0) * wfcsc[i] + ay[y] * zetaysc[i];
      psiyn[i] = -ay[y] * DIFFY1(PSIY_TERM) + ay[y] * psiy[i];
      zetayn[i] = ay[y] * V2DT2(0, 0) * wfc[i] +
                  ay[y] * 2 * VDT2(0, 0) * SCATTER(0, 0) * wfcsc[i] +
                  ay[y] * zetay[i];
    }
    if (pml_x) {
      psixnsc[i] = -ax[x] * DIFFX1(PSIXSC_TERM) + ax[x] * psixsc[i];
      zetaxnsc[i] = ax[x] * V2DT2(0, 0) * wfcsc[i] + ax[x] * zetaxsc[i];
      psixn[i] = -ax[x] * DIFFX1(PSIX_TERM) + ax[x] * psix[i];
      zetaxn[i] = ax[x] * V2DT2(0, 0) * wfc[i] +
                  ax[x] * 2 * VDT2(0, 0) * SCATTER(0, 0) * wfcsc[i] +
                  ax[x] * zetax[i];
    }
    if (v_requires_grad) {
      grad_v[i] += wfc[i] * 2 * v_shot[j] * dt2 * w_store[i] * step_ratio +
                   wfcsc[i] *
                       (2 * dt2 * scatter_shot[j] * w_store[i] +
                        2 * v_shot[j] * dt2 * wsc_store[i]) *
                       step_ratio;
    }
    if (scatter_requires_grad) {
      grad_scatter[i] += wfcsc[i] * 2 * v_shot[j] * dt2 * w_store[i] * step_ratio;
    }
  }
}

__global__ void backward_kernel_sc(
    DW_DTYPE const *__restrict const v, DW_DTYPE const *__restrict const wfcsc,
    DW_DTYPE *__restrict const wfpsc, DW_DTYPE const *__restrict const psiysc,
    DW_DTYPE const *__restrict const psixsc, DW_DTYPE *__restrict const psiynsc,
    DW_DTYPE *__restrict const psixnsc, DW_DTYPE *__restrict const zetaysc,
    DW_DTYPE *__restrict const zetaxsc, DW_DTYPE *__restrict const zetaynsc,
    DW_DTYPE *__restrict const zetaxnsc,
    DW_DTYPE const *__restrict const w_store,
    DW_DTYPE *__restrict const grad_scatter,
    DW_DTYPE const *__restrict const ay, DW_DTYPE const *__restrict const ax,
    DW_DTYPE const *__restrict const by, DW_DTYPE const *__restrict const bx,
    DW_DTYPE const *__restrict const dbydy,
    DW_DTYPE const *__restrict const dbxdx, bool const scatter_requires_grad) {
  int64_t x = blockIdx.x * blockDim.x + threadIdx.x + FD_PAD;
  int64_t y = blockIdx.y * blockDim.y + threadIdx.y + FD_PAD;
  if (y < ny - FD_PAD && x < nx - FD_PAD) {
    int64_t shot_idx = blockIdx.z * blockDim.z + threadIdx.z;
    int64_t j = y * nx + x;
    int64_t i = shot_idx * nynx + j;
    DW_DTYPE const *__restrict const v_shot = v_batched ? v + shot_idx * nynx : v;
    bool pml_y = y < pml_y0 || y >= pml_y1;
    bool pml_x = x < pml_x0 || x >= pml_x1;
    wfpsc[i] = (pml_y ? -DIFFY1(UTSC_TERMY1) + DIFFY2(UTSC_TERMY2)
                      : DIFFY2(V2DT2_WFCSC)) +
               (pml_x ? -DIFFX1(UTSC_TERMX1) + DIFFX2(UTSC_TERMX2)
                      : DIFFX2(V2DT2_WFCSC)) +
               2 * wfcsc[i] - wfpsc[i];
    if (pml_y) {
      psiynsc[i] = -ay[y] * DIFFY1(PSIYSC_TERM) + ay[y] * psiysc[i];
      zetaynsc[i] = ay[y] * V2DT2(0, 0) * wfcsc[i] + ay[y] * zetaysc[i];
    }
    if (pml_x) {
      psixnsc[i] = -ax[x] * DIFFX1(PSIXSC_TERM) + ax[x] * psixsc[i];
      zetaxnsc[i] = ax[x] * V2DT2(0, 0) * wfcsc[i] + ax[x] * zetaxsc[i];
    }
    if (scatter_requires_grad) {
      grad_scatter[i] += wfcsc[i] * 2 * v_shot[j] * dt2 * w_store[i] * step_ratio;
    }
  }
}

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

inline unsigned int ceil_div(unsigned int numerator, unsigned int denominator) {
  return (numerator + denominator - 1) / denominator;
}

void set_config(
    DW_DTYPE const dt2_h, DW_DTYPE const rdy_h, DW_DTYPE const rdx_h,
    DW_DTYPE const rdy2_h, DW_DTYPE const rdx2_h, int64_t const n_shots_h,
    int64_t const ny_h, int64_t const nx_h, int64_t const n_sources_per_shot_h,
    int64_t const n_sourcessc_per_shot_h, int64_t const n_receivers_per_shot_h,
    int64_t const n_receiverssc_per_shot_h, int64_t const step_ratio_h,
    int64_t const pml_y0_h, int64_t const pml_y1_h, int64_t const pml_x0_h,
    int64_t const pml_x1_h, bool const v_batched_h, bool const scatter_batched_h) {
  int64_t const nynx_h = ny_h * nx_h;
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dt2), &dt2_h, sizeof(DW_DTYPE)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(rdy), &rdy_h, sizeof(DW_DTYPE)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(rdx), &rdx_h, sizeof(DW_DTYPE)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(rdy2), &rdy2_h, sizeof(DW_DTYPE)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(rdx2), &rdx2_h, sizeof(DW_DTYPE)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(n_shots), &n_shots_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ny), &ny_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(nx), &nx_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(nynx), &nynx_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(n_sources_per_shot), &n_sources_per_shot_h,
                               sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(n_sourcessc_per_shot), &n_sourcessc_per_shot_h,
                               sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(n_receivers_per_shot), &n_receivers_per_shot_h,
                               sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(n_receiverssc_per_shot),
                               &n_receiverssc_per_shot_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(step_ratio), &step_ratio_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(pml_y0), &pml_y0_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(pml_y1), &pml_y1_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(pml_x0), &pml_x0_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(pml_x1), &pml_x1_h, sizeof(int64_t)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(v_batched), &v_batched_h, sizeof(bool)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(scatter_batched), &scatter_batched_h, sizeof(bool)));
}

}  // namespace

extern "C"
#ifdef _WIN32
    __declspec(dllexport)
#endif
        void FUNC(forward)(
            DW_DTYPE const *__restrict const v,
            DW_DTYPE const *__restrict const scatter,
            DW_DTYPE const *__restrict const f,
            DW_DTYPE const *__restrict const fsc,
            DW_DTYPE *__restrict const wfc, DW_DTYPE *__restrict const wfp,
            DW_DTYPE *__restrict const psiy, DW_DTYPE *__restrict const psix,
            DW_DTYPE *__restrict const psiyn, DW_DTYPE *__restrict const psixn,
            DW_DTYPE *__restrict const zetay, DW_DTYPE *__restrict const zetax,
            DW_DTYPE *__restrict const wfcsc, DW_DTYPE *__restrict const wfpsc,
            DW_DTYPE *__restrict const psiysc,
            DW_DTYPE *__restrict const psixsc,
            DW_DTYPE *__restrict const psiynsc,
            DW_DTYPE *__restrict const psixnsc,
            DW_DTYPE *__restrict const zetaysc,
            DW_DTYPE *__restrict const zetaxsc,
            DW_DTYPE *__restrict const w_store,
            DW_DTYPE *__restrict const wsc_store, DW_DTYPE *__restrict const r,
            DW_DTYPE *__restrict const rsc, DW_DTYPE const *__restrict const ay,
            DW_DTYPE const *__restrict const ax,
            DW_DTYPE const *__restrict const by,
            DW_DTYPE const *__restrict const bx,
            DW_DTYPE const *__restrict const dbydy,
            DW_DTYPE const *__restrict const dbxdx,
            int64_t const *__restrict const sources_i,
            int64_t const *__restrict const receivers_i,
            int64_t const *__restrict const receiverssc_i, DW_DTYPE const rdy_h,
            DW_DTYPE const rdx_h, DW_DTYPE const rdy2_h, DW_DTYPE const rdx2_h,
            DW_DTYPE const dt2_h, int64_t const nt, int64_t const n_shots_h,
            int64_t const ny_h, int64_t const nx_h,
            int64_t const n_sources_per_shot_h,
            int64_t const n_receivers_per_shot_h,
            int64_t const n_receiverssc_per_shot_h, int64_t const step_ratio_h,
            bool const v_requires_grad, bool const scatter_requires_grad,
            bool const v_batched_h, bool const scatter_batched_h, int64_t const start_t,
            int64_t const pml_y0_h, int64_t const pml_y1_h,
            int64_t const pml_x0_h, int64_t const pml_x1_h,
            int64_t const device) {

  dim3 dimBlock(32, 32, 1);
  unsigned int gridx = ceil_div(nx_h - 2 * FD_PAD, dimBlock.x);
  unsigned int gridy = ceil_div(ny_h - 2 * FD_PAD, dimBlock.y);
  unsigned int gridz = ceil_div(n_shots_h, dimBlock.z);
  dim3 dimGrid(gridx, gridy, gridz);
  dim3 dimBlock_sources(32, 1, 1);
  unsigned int gridx_sources =
      ceil_div(n_sources_per_shot_h, dimBlock_sources.x);
  unsigned int gridy_sources = ceil_div(n_shots_h, dimBlock_sources.y);
  unsigned int gridz_sources = 1;
  dim3 dimGrid_sources(gridx_sources, gridy_sources, gridz_sources);
  dim3 dimBlock_receivers(32, 1, 1);
  unsigned int gridx_receivers =
      ceil_div(n_receivers_per_shot_h, dimBlock_receivers.x);
  unsigned int gridy_receivers = ceil_div(n_shots_h, dimBlock_receivers.y);
  unsigned int gridz_receivers = 1;
  dim3 dimGrid_receivers(gridx_receivers, gridy_receivers, gridz_receivers);
  dim3 dimBlock_receiverssc(32, 1, 1);
  unsigned int gridx_receiverssc =
      ceil_div(n_receiverssc_per_shot_h, dimBlock_receiverssc.x);
  unsigned int gridy_receiverssc = ceil_div(n_shots_h, dimBlock_receiverssc.y);
  unsigned int gridz_receiverssc = 1;
  dim3 dimGrid_receiverssc(gridx_receiverssc, gridy_receiverssc,
                           gridz_receiverssc);

  int64_t t;
  gpuErrchk(hipSetDevice(device));
  set_config(dt2_h, rdy_h, rdx_h, rdy2_h, rdx2_h, n_shots_h, ny_h, nx_h,
             n_sources_per_shot_h, n_sources_per_shot_h, n_receivers_per_shot_h,
             n_receiverssc_per_shot_h, step_ratio_h, pml_y0_h, pml_y1_h,
             pml_x0_h, pml_x1_h, v_batched_h, scatter_batched_h);
  for (t = 0; t < nt; ++t) {
    if (t & 1) {
      forward_kernel<<<dimGrid, dimBlock>>>(
          v, scatter, wfp, wfc, psiyn, psixn, psiy, psix, zetay, zetax, wfpsc,
          wfcsc, psiynsc, psixnsc, psiysc, psixsc, zetaysc, zetaxsc,
          w_store + (t / step_ratio_h) * ny_h * nx_h * n_shots_h,
          wsc_store + (t / step_ratio_h) * ny_h * nx_h * n_shots_h, ay, ax, by,
          bx, dbydy, dbxdx, v_requires_grad && (((t + start_t) % step_ratio_h) == 0),
          scatter_requires_grad && (((t + start_t) % step_ratio_h) == 0));
      CHECK_KERNEL_ERROR
      if (n_sources_per_shot_h > 0) {
        add_sources_both<<<dimGrid_sources, dimBlock_sources>>>(
            wfc, wfcsc, f + t * n_shots_h * n_sources_per_shot_h,
            fsc + t * n_shots_h * n_sources_per_shot_h, sources_i);
        CHECK_KERNEL_ERROR
      }
      if (n_receivers_per_shot_h > 0) {
        record_receivers<<<dimGrid_receivers, dimBlock_receivers>>>(
            r + t * n_shots_h * n_receivers_per_shot_h, wfp, receivers_i);
        CHECK_KERNEL_ERROR
      }
      if (n_receiverssc_per_shot_h > 0) {
        record_receiverssc<<<dimGrid_receiverssc, dimBlock_receiverssc>>>(
            rsc + t * n_shots_h * n_receiverssc_per_shot_h, wfpsc,
            receiverssc_i);
        CHECK_KERNEL_ERROR
      }

    } else {
      forward_kernel<<<dimGrid, dimBlock>>>(
          v, scatter, wfc, wfp, psiy, psix, psiyn, psixn, zetay, zetax, wfcsc,
          wfpsc, psiysc, psixsc, psiynsc, psixnsc, zetaysc, zetaxsc,
          w_store + (t / step_ratio_h) * ny_h * nx_h * n_shots_h,
          wsc_store + (t / step_ratio_h) * ny_h * nx_h * n_shots_h, ay, ax, by,
          bx, dbydy, dbxdx, v_requires_grad && (((t + start_t) % step_ratio_h) == 0),
          scatter_requires_grad && (((t + start_t) % step_ratio_h) == 0));
      CHECK_KERNEL_ERROR
      if (n_sources_per_shot_h > 0) {
        add_sources_both<<<dimGrid_sources, dimBlock_sources>>>(
            wfp, wfpsc, f + t * n_shots_h * n_sources_per_shot_h,
            fsc + t * n_shots_h * n_sources_per_shot_h, sources_i);
        CHECK_KERNEL_ERROR
      }
      if (n_receivers_per_shot_h > 0) {
        record_receivers<<<dimGrid_receivers, dimBlock_receivers>>>(
            r + t * n_shots_h * n_receivers_per_shot_h, wfc, receivers_i);
        CHECK_KERNEL_ERROR
      }
      if (n_receiverssc_per_shot_h > 0) {
        record_receiverssc<<<dimGrid_receiverssc, dimBlock_receiverssc>>>(
            rsc + t * n_shots_h * n_receiverssc_per_shot_h, wfcsc,
            receiverssc_i);
        CHECK_KERNEL_ERROR
      }
    }
  }
}

extern "C"
#ifdef _WIN32
    __declspec(dllexport)
#endif
        void FUNC(backward)(
            DW_DTYPE const *__restrict const v,
            DW_DTYPE const *__restrict const scatter,
            DW_DTYPE const *__restrict const grad_r,
            DW_DTYPE const *__restrict const grad_rsc,
            DW_DTYPE *__restrict const wfc, DW_DTYPE *__restrict const wfp,
            DW_DTYPE *__restrict const psiy, DW_DTYPE *__restrict const psix,
            DW_DTYPE *__restrict const psiyn, DW_DTYPE *__restrict const psixn,
            DW_DTYPE *__restrict const zetay, DW_DTYPE *__restrict const zetax,
            DW_DTYPE *__restrict const zetayn,
            DW_DTYPE *__restrict const zetaxn, DW_DTYPE *__restrict const wfcsc,
            DW_DTYPE *__restrict const wfpsc, DW_DTYPE *__restrict const psiysc,
            DW_DTYPE *__restrict const psixsc,
            DW_DTYPE *__restrict const psiynsc,
            DW_DTYPE *__restrict const psixnsc,
            DW_DTYPE *__restrict const zetaysc,
            DW_DTYPE *__restrict const zetaxsc,
            DW_DTYPE *__restrict const zetaynsc,
            DW_DTYPE *__restrict const zetaxnsc,
            DW_DTYPE const *__restrict const w_store,
            DW_DTYPE const *__restrict const wsc_store,
            DW_DTYPE *__restrict const grad_f,
            DW_DTYPE *__restrict const grad_fsc,
            DW_DTYPE *__restrict const grad_v,
            DW_DTYPE *__restrict const grad_scatter,
            DW_DTYPE *__restrict const grad_v_shot,
            DW_DTYPE *__restrict const grad_scatter_shot,
            DW_DTYPE const *__restrict const ay,
            DW_DTYPE const *__restrict const ax,
            DW_DTYPE const *__restrict const by,
            DW_DTYPE const *__restrict const bx,
            DW_DTYPE const *__restrict const dbydy,
            DW_DTYPE const *__restrict const dbxdx,
            int64_t const *__restrict const sources_i,
            int64_t const *__restrict const receivers_i,
            int64_t const *__restrict const receiverssc_i, DW_DTYPE const rdy_h,
            DW_DTYPE const rdx_h, DW_DTYPE const rdy2_h, DW_DTYPE const rdx2_h,
            DW_DTYPE const dt2_h, int64_t const nt, int64_t const n_shots_h,
            int64_t const ny_h, int64_t const nx_h,
            int64_t const n_sources_per_shot_h,
            int64_t const n_sourcessc_per_shot_h,
            int64_t const n_receivers_per_shot_h,
            int64_t const n_receiverssc_per_shot_h, int64_t const step_ratio_h,
            bool const v_requires_grad, bool const scatter_requires_grad,
            bool const v_batched_h, bool const scatter_batched_h, int64_t const start_t,
            int64_t const pml_y0_h, int64_t const pml_y1_h,
            int64_t const pml_x0_h, int64_t const pml_x1_h,
            int64_t const device) {

  dim3 dimBlock(32, 8, 1);
  unsigned int gridx = ceil_div(nx_h - 2 * FD_PAD, dimBlock.x);
  unsigned int gridy = ceil_div(ny_h - 2 * FD_PAD, dimBlock.y);
  unsigned int gridz = ceil_div(n_shots_h, dimBlock.z);
  dim3 dimGrid(gridx, gridy, gridz);
  dim3 dimBlock_sources(32, 1, 1);
  unsigned int gridx_sources =
      ceil_div(n_sources_per_shot_h, dimBlock_sources.x);
  unsigned int gridy_sources = ceil_div(n_shots_h, dimBlock_sources.y);
  unsigned int gridz_sources = 1;
  dim3 dimGrid_sources(gridx_sources, gridy_sources, gridz_sources);
  dim3 dimBlock_sourcessc(32, 1, 1);
  unsigned int gridx_sourcessc =
      ceil_div(n_sourcessc_per_shot_h, dimBlock_sourcessc.x);
  unsigned int gridy_sourcessc = ceil_div(n_shots_h, dimBlock_sourcessc.y);
  unsigned int gridz_sourcessc = 1;
  dim3 dimGrid_sourcessc(gridx_sourcessc, gridy_sourcessc, gridz_sourcessc);
  dim3 dimBlock_receivers(32, 1, 1);
  unsigned int gridx_receivers =
      ceil_div(n_receivers_per_shot_h, dimBlock_receivers.x);
  unsigned int gridy_receivers = ceil_div(n_shots_h, dimBlock_receivers.y);
  unsigned int gridz_receivers = 1;
  dim3 dimGrid_receivers(gridx_receivers, gridy_receivers, gridz_receivers);
  dim3 dimBlock_receiverssc(32, 1, 1);
  unsigned int gridx_receiverssc =
      ceil_div(n_receiverssc_per_shot_h, dimBlock_receiverssc.x);
  unsigned int gridy_receiverssc = ceil_div(n_shots_h, dimBlock_receiverssc.y);
  unsigned int gridz_receiverssc = 1;
  dim3 dimGrid_receiverssc(gridx_receiverssc, gridy_receiverssc,
                           gridz_receiverssc);
  dim3 dimBlock_combine(32, 32, 1);
  unsigned int gridx_combine = ceil_div(nx_h - 2 * FD_PAD, dimBlock_combine.x);
  unsigned int gridy_combine = ceil_div(ny_h - 2 * FD_PAD, dimBlock_combine.y);
  unsigned int gridz_combine = 1;
  dim3 dimGrid_combine(gridx_combine, gridy_combine, gridz_combine);
  int64_t t;
  gpuErrchk(hipSetDevice(device));
  set_config(dt2_h, rdy_h, rdx_h, rdy2_h, rdx2_h, n_shots_h, ny_h, nx_h,
             n_sources_per_shot_h, n_sourcessc_per_shot_h,
             n_receivers_per_shot_h, n_receiverssc_per_shot_h, step_ratio_h,
             pml_y0_h, pml_y1_h, pml_x0_h, pml_x1_h, v_batched_h, scatter_batched_h);
  for (t = nt - 1; t >= 0; --t) {
    if ((nt - 1 - t) & 1) {
      if (n_sources_per_shot_h > 0) {
        record_adjoint_receivers<<<dimGrid_sources, dimBlock_sources>>>(
            grad_f + t * n_shots_h * n_sources_per_shot_h, wfp, sources_i);
        CHECK_KERNEL_ERROR
      }
      if (n_sourcessc_per_shot_h > 0) {
        record_adjoint_receiverssc<<<dimGrid_sourcessc, dimBlock_sourcessc>>>(
            grad_fsc + t * n_shots_h * n_sourcessc_per_shot_h, wfpsc,
            sources_i);
        CHECK_KERNEL_ERROR
      }
      backward_kernel<<<dimGrid, dimBlock>>>(
          v, scatter, wfp, wfc, psiyn, psixn, psiy, psix, zetayn, zetaxn, zetay,
          zetax, wfpsc, wfcsc, psiynsc, psixnsc, psiysc, psixsc, zetaynsc,
          zetaxnsc, zetaysc, zetaxsc,
          w_store + (t / step_ratio_h) * n_shots_h * ny_h * nx_h,
          wsc_store + (t / step_ratio_h) * n_shots_h * ny_h * nx_h, grad_v_shot,
          grad_scatter_shot, ay, ax, by, bx, dbydy, dbxdx,
          v_requires_grad && (((t + start_t) % step_ratio_h) == 0),
          scatter_requires_grad && (((t + start_t) % step_ratio_h) == 0));
      CHECK_KERNEL_ERROR
      if (n_receivers_per_shot_h > 0) {
        add_adjoint_sources<<<dimGrid_receivers, dimBlock_receivers>>>(
            wfc, grad_r + t * n_shots_h * n_receivers_per_shot_h, receivers_i);
        CHECK_KERNEL_ERROR
      }
      if (n_receiverssc_per_shot_h > 0) {
        add_adjoint_sourcessc<<<dimGrid_receiverssc, dimBlock_receiverssc>>>(
            wfcsc, grad_rsc + t * n_shots_h * n_receiverssc_per_shot_h,
            receiverssc_i);
        CHECK_KERNEL_ERROR
      }
    } else {
      if (n_sources_per_shot_h > 0) {
        record_adjoint_receivers<<<dimGrid_sources, dimBlock_sources>>>(
            grad_f + t * n_shots_h * n_sources_per_shot_h, wfc, sources_i);
        CHECK_KERNEL_ERROR
      }
      if (n_sourcessc_per_shot_h > 0) {
        record_adjoint_receiverssc<<<dimGrid_sourcessc, dimBlock_sourcessc>>>(
            grad_fsc + t * n_shots_h * n_sourcessc_per_shot_h, wfcsc,
            sources_i);
        CHECK_KERNEL_ERROR
      }
      backward_kernel<<<dimGrid, dimBlock>>>(
          v, scatter, wfc, wfp, psiy, psix, psiyn, psixn, zetay, zetax, zetayn,
          zetaxn, wfcsc, wfpsc, psiysc, psixsc, psiynsc, psixnsc, zetaysc,
          zetaxsc, zetaynsc, zetaxnsc,
          w_store + (t / step_ratio_h) * n_shots_h * ny_h * nx_h,
          wsc_store + (t / step_ratio_h) * n_shots_h * ny_h * nx_h, grad_v_shot,
          grad_scatter_shot, ay, ax, by, bx, dbydy, dbxdx,
          v_requires_grad && (((t + start_t) % step_ratio_h) == 0),
          scatter_requires_grad && (((t + start_t) % step_ratio_h) == 0));
      CHECK_KERNEL_ERROR
      if (n_receivers_per_shot_h > 0) {
        add_adjoint_sources<<<dimGrid_receivers, dimBlock_receivers>>>(
            wfp, grad_r + t * n_shots_h * n_receivers_per_shot_h, receivers_i);
        CHECK_KERNEL_ERROR
      }
      if (n_receiverssc_per_shot_h > 0) {
        add_adjoint_sourcessc<<<dimGrid_receiverssc, dimBlock_receiverssc>>>(
            wfpsc, grad_rsc + t * n_shots_h * n_receiverssc_per_shot_h,
            receiverssc_i);
        CHECK_KERNEL_ERROR
      }
    }
  }
  if (v_requires_grad && !v_batched_h && n_shots_h > 1) {
    combine_grad<<<dimGrid_combine, dimBlock_combine>>>(grad_v, grad_v_shot);
    CHECK_KERNEL_ERROR
  }
  if (scatter_requires_grad && !scatter_batched_h && n_shots_h > 1) {
    combine_grad<<<dimGrid_combine, dimBlock_combine>>>(grad_scatter,
                                                        grad_scatter_shot);
    CHECK_KERNEL_ERROR
  }
}

extern "C"
#ifdef _WIN32
    __declspec(dllexport)
#endif
        void FUNC(backward_sc)(
            DW_DTYPE const *__restrict const v,
            DW_DTYPE const *__restrict const grad_rsc,
            DW_DTYPE *__restrict const wfcsc, DW_DTYPE *__restrict const wfpsc,
            DW_DTYPE *__restrict const psiysc,
            DW_DTYPE *__restrict const psixsc,
            DW_DTYPE *__restrict const psiynsc,
            DW_DTYPE *__restrict const psixnsc,
            DW_DTYPE *__restrict const zetaysc,
            DW_DTYPE *__restrict const zetaxsc,
            DW_DTYPE *__restrict const zetaynsc,
            DW_DTYPE *__restrict const zetaxnsc,
            DW_DTYPE const *__restrict const w_store,
            DW_DTYPE *__restrict const grad_fsc,
            DW_DTYPE *__restrict const grad_scatter,
            DW_DTYPE *__restrict const grad_scatter_shot,
            DW_DTYPE const *__restrict const ay,
            DW_DTYPE const *__restrict const ax,
            DW_DTYPE const *__restrict const by,
            DW_DTYPE const *__restrict const bx,
            DW_DTYPE const *__restrict const dbydy,
            DW_DTYPE const *__restrict const dbxdx,
            int64_t const *__restrict const sources_i,
            int64_t const *__restrict const receiverssc_i, DW_DTYPE const rdy_h,
            DW_DTYPE const rdx_h, DW_DTYPE const rdy2_h, DW_DTYPE const rdx2_h,
            DW_DTYPE const dt2_h, int64_t const nt, int64_t const n_shots_h,
            int64_t const ny_h, int64_t const nx_h,
            int64_t const n_sourcessc_per_shot_h,
            int64_t const n_receiverssc_per_shot_h, int64_t const step_ratio_h,
            bool const scatter_requires_grad,
            bool const v_batched_h, bool const scatter_batched_h, int64_t const start_t,
	    int64_t const pml_y0_h,
            int64_t const pml_y1_h, int64_t const pml_x0_h,
            int64_t const pml_x1_h, int64_t const device) {

  dim3 dimBlock(32, 16, 1);
  unsigned int gridx = ceil_div(nx_h - 2 * FD_PAD, dimBlock.x);
  unsigned int gridy = ceil_div(ny_h - 2 * FD_PAD, dimBlock.y);
  unsigned int gridz = ceil_div(n_shots_h, dimBlock.z);
  dim3 dimGrid(gridx, gridy, gridz);
  dim3 dimBlock_sourcessc(32, 1, 1);
  unsigned int gridx_sourcessc =
      ceil_div(n_sourcessc_per_shot_h, dimBlock_sourcessc.x);
  unsigned int gridy_sourcessc = ceil_div(n_shots_h, dimBlock_sourcessc.y);
  unsigned int gridz_sourcessc = 1;
  dim3 dimGrid_sourcessc(gridx_sourcessc, gridy_sourcessc, gridz_sourcessc);
  dim3 dimBlock_receiverssc(32, 1, 1);
  unsigned int gridx_receiverssc =
      ceil_div(n_receiverssc_per_shot_h, dimBlock_receiverssc.x);
  unsigned int gridy_receiverssc = ceil_div(n_shots_h, dimBlock_receiverssc.y);
  unsigned int gridz_receiverssc = 1;
  dim3 dimGrid_receiverssc(gridx_receiverssc, gridy_receiverssc,
                           gridz_receiverssc);
  dim3 dimBlock_combine(32, 32, 1);
  unsigned int gridx_combine = ceil_div(nx_h - 2 * FD_PAD, dimBlock_combine.x);
  unsigned int gridy_combine = ceil_div(ny_h - 2 * FD_PAD, dimBlock_combine.y);
  unsigned int gridz_combine = 1;
  dim3 dimGrid_combine(gridx_combine, gridy_combine, gridz_combine);
  int64_t t;
  gpuErrchk(hipSetDevice(device));
  set_config(dt2_h, rdy_h, rdx_h, rdy2_h, rdx2_h, n_shots_h, ny_h, nx_h,
             n_sourcessc_per_shot_h, n_sourcessc_per_shot_h,
             n_receiverssc_per_shot_h, n_receiverssc_per_shot_h, step_ratio_h,
             pml_y0_h, pml_y1_h, pml_x0_h, pml_x1_h, v_batched_h, scatter_batched_h);
  for (t = nt - 1; t >= 0; --t) {
    if ((nt - 1 - t) & 1) {
      if (n_sourcessc_per_shot_h > 0) {
        record_adjoint_receiverssc<<<dimGrid_sourcessc, dimBlock_sourcessc>>>(
            grad_fsc + t * n_shots_h * n_sourcessc_per_shot_h, wfpsc,
            sources_i);
        CHECK_KERNEL_ERROR
      }
      backward_kernel_sc<<<dimGrid, dimBlock>>>(
          v, wfpsc, wfcsc, psiynsc, psixnsc, psiysc, psixsc, zetaynsc, zetaxnsc,
          zetaysc, zetaxsc,
          w_store + (t / step_ratio_h) * n_shots_h * ny_h * nx_h,
          grad_scatter_shot, ay, ax, by, bx, dbydy, dbxdx,
          scatter_requires_grad && (((t + start_t) % step_ratio_h) == 0));
      CHECK_KERNEL_ERROR
      if (n_receiverssc_per_shot_h > 0) {
        add_adjoint_sourcessc<<<dimGrid_receiverssc, dimBlock_receiverssc>>>(
            wfcsc, grad_rsc + t * n_shots_h * n_receiverssc_per_shot_h,
            receiverssc_i);
        CHECK_KERNEL_ERROR
      }
    } else {
      if (n_sourcessc_per_shot_h > 0) {
        record_adjoint_receiverssc<<<dimGrid_sourcessc, dimBlock_sourcessc>>>(
            grad_fsc + t * n_shots_h * n_sourcessc_per_shot_h, wfcsc,
            sources_i);
        CHECK_KERNEL_ERROR
      }
      backward_kernel_sc<<<dimGrid, dimBlock>>>(
          v, wfcsc, wfpsc, psiysc, psixsc, psiynsc, psixnsc, zetaysc, zetaxsc,
          zetaynsc, zetaxnsc,
          w_store + (t / step_ratio_h) * n_shots_h * ny_h * nx_h,
          grad_scatter_shot, ay, ax, by, bx, dbydy, dbxdx,
          scatter_requires_grad && (((t + start_t) % step_ratio_h) == 0));
      CHECK_KERNEL_ERROR
      if (n_receiverssc_per_shot_h > 0) {
        add_adjoint_sourcessc<<<dimGrid_receiverssc, dimBlock_receiverssc>>>(
            wfpsc, grad_rsc + t * n_shots_h * n_receiverssc_per_shot_h,
            receiverssc_i);
        CHECK_KERNEL_ERROR
      }
    }
  }
  if (scatter_requires_grad && !scatter_batched_h && n_shots_h > 1) {
    combine_grad<<<dimGrid_combine, dimBlock_combine>>>(grad_scatter,
                                                        grad_scatter_shot);
    CHECK_KERNEL_ERROR
  }
}
